#include "hip/hip_runtime.h"
#include "assignment_kernel.hpp"

#include ""

using index_vector_t = recap::cuda::vector4<recap::cuda::resistance_t>;

/** Convert linear index to table index (the table is 4 dimensional - one for each type of resistance)
 * 
 * @param idx Linear index
 * @param table_dim Dimensions of the table
 * 
 * @returns index for each dimension in the table
 */
__device__ index_vector_t index_to_vector(int idx, index_vector_t table_dim)
{
    index_vector_t result;
    result.w = idx % table_dim.w;
    idx /= table_dim.w;
    result.z = idx % table_dim.z;
    idx /= table_dim.z;
    result.y = idx % table_dim.y;
    idx /= table_dim.y;
    result.x = idx;

    return result;
}

/** Convert table indices into a single linear index
 * 
 * @param indices Indices for each dimension
 * @param table_dim Dimensions of the table
 * 
 * @returns linear index
 */
__device__ int vector_to_index(index_vector_t indices, index_vector_t table_dim)
{
    int index = indices.x;
    index = index * table_dim.y + indices.y;
    index = index * table_dim.z + indices.z;
    index = index * table_dim.w + indices.w;
    return index;
}

/** Subtract @p rhs from @p lhs
 * 
 * If the result is negative, it's clamped to 0.
 * 
 * @param lhs Left-hand side of the expression
 * @param rhs Right-hand side of the expression
 * 
 * @returns result
 */
__device__ index_vector_t index_vector_sub(index_vector_t lhs, index_vector_t rhs)
{
    index_vector_t result;
    result.x = max(lhs.x - (int)rhs.x, 0);
    result.y = max(lhs.y - (int)rhs.y, 0);
    result.z = max(lhs.z - (int)rhs.z, 0);
    result.w = max(lhs.w - (int)rhs.w, 0);
    return result;
}

/** Find best assignment 
 * 
 * @param input Input parameters
 * @param output Output parameters
 */
__global__ void assignment_kernel(
    const recap::cuda::input_data input, 
    recap::cuda::output_data output)
{
    auto current_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (current_index >= input.table_size)
    {
        return;
    }

    // find resistances at current index
    auto current_resist = index_to_vector(current_index, input.table_dim);
    auto current_cost = output.best_cost[current_index];

    // find resistances if we use assigned recipe
    auto prev_resist = index_vector_sub(current_resist, input.recipe_resist);
    auto prev_index = vector_to_index(prev_resist, input.table_dim);
    auto prev_cost = input.best_cost[prev_index];

    // if using the recipe is better then current best solution, update the solution
    if (current_cost > prev_cost + input.recipe_cost)
    {
        output.best_cost[current_index] = prev_cost + input.recipe_cost;

        for (int i = 0; i < recap::cuda::MAX_SLOT_COUNT; ++i)
        {
            auto current_slot_index = current_index * recap::cuda::MAX_SLOT_COUNT + i;
            auto prev_slot_index = prev_index * recap::cuda::MAX_SLOT_COUNT + i;
            output.best_assignment[current_slot_index] = input.best_assignment[prev_slot_index];
        }
        output.best_assignment[current_index * recap::cuda::MAX_SLOT_COUNT + input.slot_index] = input.recipe_index;
    }
}

void recap::cuda::run_assignment_kernel(const input_data input, output_data output)
{
    constexpr int block_size = 256;

    ::assignment_kernel<<<(input.table_size + block_size - 1) / block_size, block_size>>>(input, output);
}